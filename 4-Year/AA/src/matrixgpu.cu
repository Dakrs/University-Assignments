#include "hip/hip_runtime.h"
#include "matrixgpu.cuh"

hipEvent_t start, stop, transferTodevice, transferTohost, startTohost;

float datatransfer_time;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void measuse_data_transfer_time_to_device(void){
    hipEventRecord(transferTodevice);

    hipEventSynchronize(transferTodevice);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, transferTodevice);
    datatransfer_time += milliseconds;
}

void startTransferToHost(void){
    hipEventRecord(startTohost);
}

void stopTransferToHost(void){
    hipEventRecord(transferTohost);

    hipEventSynchronize(transferTohost);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, startTohost, transferTohost);
    datatransfer_time += milliseconds;
}

void startKernelTime (void) {
	hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&transferTodevice);
    hipEventCreate(&transferTohost);
    hipEventCreate(&startTohost);


	hipEventRecord(start);
}

void stopKernelTime (void) {
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	cout << milliseconds << " ms have elapsed for the CUDA execution" << endl;
}

void checkCUDAError (const char *msg) {
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		cerr << "Cuda error: " << msg << ", " << hipGetErrorString( err) << endl;
		exit(-1);
	}
}


__global__
void block_dot_product_v1(float * A,float * B,float * C, int N){
    int rowBlock = blockIdx.y*blockDim.y + threadIdx.y;
    int colBlock = blockIdx.x*blockDim.x + threadIdx.x;

    float result = 0;
    __shared__ float sA[TILE_SIZE][TILE_SIZE];
    for(int i = 0; i < N; i += TILE_SIZE){
        sA[threadIdx.y][threadIdx.x] = A[rowBlock*N + i + threadIdx.x];

        __syncthreads();
        for(int k = 0; k < TILE_SIZE; k++){
            result += sA[threadIdx.y][k] * B[(i+k)*N + colBlock];
        }
        __syncthreads();
    }
    C[rowBlock*N + colBlock] = result;
}


__global__
void block_dot_product_v2(float * A,float * B,float * C, int N){
    int rowBlock = blockIdx.y*blockDim.y + threadIdx.y;
    int colBlock = blockIdx.x*blockDim.x + threadIdx.x;

    float result = 0;
    __shared__ float sA[TILE_SIZE][TILE_SIZE];
    __shared__ float sB[TILE_SIZE][TILE_SIZE];
    for(int i = 0; i < N; i += TILE_SIZE){
        sA[threadIdx.y][threadIdx.x] = A[rowBlock*N + i + threadIdx.x];
        sB[threadIdx.y][threadIdx.x] = B[(i+threadIdx.y)*N + colBlock];
        __syncthreads();
        for(int k = 0; k < TILE_SIZE; k++){
            result += sA[threadIdx.y][k] * sB[k][threadIdx.x];
        }
        __syncthreads();
    }
    C[rowBlock*N + colBlock] = result;
}

__global__
void block_dot_product(float * A,float * B,float * C, int N){
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int column = blockIdx.x*blockDim.x + threadIdx.x;

    float result = 0.0;

    for(int i = 0; i < N; i++)
        result += A[row*N + i] * B[i*N + column];
    C[row*N + column] = result;
}

void dot_product_gpu(float * A,float * B,float * C, int N){
    float * matA, * matB, * output;
    int bytes = N*N*sizeof(float);

    datatransfer_time = 0.0;

    hipMalloc((void**) &matA, bytes);
    hipMalloc((void**) &matB, bytes);
    hipMalloc((void**) &output, bytes);

    checkCUDAError("mem allocation");

    startKernelTime();

    hipMemcpy(matA,A,bytes,hipMemcpyHostToDevice);
    hipMemcpy(matB,B,bytes,hipMemcpyHostToDevice);
    hipMemset(output,0.0,bytes);

    measuse_data_transfer_time_to_device();

    checkCUDAError("mem set");

    dim3 threadsPerBlock(TILE_SIZE,TILE_SIZE);
    dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);

    block_dot_product_v2 <<< numBlocks , threadsPerBlock >>> (matA,matB,output,N);

    hipDeviceSynchronize();

    startTransferToHost();

    hipMemcpy(C,output,bytes,hipMemcpyDeviceToHost);

    stopTransferToHost();

    stopKernelTime();

    cout << "Data Transfer time: " << datatransfer_time << endl;

    hipFree(matA);
    hipFree(matB);
    hipFree(output);

    //bool check = checksum(A,C,N);
    //cout << check << endl;
}